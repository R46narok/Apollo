#include "memory.cuh"
#include "stdio.h"

void* allocate_global_memory(int iBytes)
{
   // printf("Allocating %d bytes\n", iBytes);
    void* ptr;
    hipMalloc(&ptr, iBytes);
    return ptr;
}

void destroy_global_memory(void* ptr)
{
    //printf("Deallocating bytes\n");
    hipFree(ptr);
}

void copy_host_to_device(void* pSrc, void* pDst, int iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyHostToDevice);
}

void copy_device_to_host(void* pSrc, void* pDst, int iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyDeviceToHost);
}

void copy_device_to_device(void* pSrc, void* pDst, int iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyDeviceToDevice);
}

void device_memset(void* pDst, int iLength, int value)
{
    hipMemset(pDst, value, iLength);
}
