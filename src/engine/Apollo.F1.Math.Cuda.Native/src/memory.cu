//
// Created by Acer on 2.7.2022 г..
//

#include "memory.cuh"

void* allocate_vram(int bytes)
{
    void* ptr;
    hipMalloc(&ptr, bytes);
    return ptr;
}

void destroy_vram(void* ptr)
{
    if (ptr != nullptr)
        hipFree(ptr);
}

void copy_host_to_device(void* src, void* dst, int length)
{
    hipMemcpy(dst, src, length, hipMemcpyHostToDevice);
}

void copy_device_to_host(void* src, void* dst, int length)
{
    hipMemcpy(dst, src, length, hipMemcpyDeviceToHost);
}