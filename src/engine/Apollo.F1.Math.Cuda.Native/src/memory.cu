#include "memory.cuh"

void* allocate_global_memory(int iBytes)
{
    void* ptr;
    hipMalloc(&ptr, iBytes);
    return ptr;
}

void destroy_global_memory(void* ptr)
{
    if (ptr != nullptr) hipFree(ptr);
}

void copy_host_to_device(void* pSrc, void* pDst, int iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyHostToDevice);
}

void copy_device_to_host(void* pSrc, void* pDst, int iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyDeviceToHost);
}

void copy_device_to_device(void* pSrc, void* pDst, int iLength)
{
    hipMemcpy(pDst, pSrc, iLength, hipMemcpyDeviceToDevice);
}

void device_memset(void* pDst, int iLength, int value)
{
    hipMemset(pDst, value, iLength);
}
