#include "hip/hip_runtime.h"
#include "multiplication.cuh"

#include "nvtx3/nvToolsExt.h"
#define BLOCK_SIZE 8

__global__ void multiply_kernel(double* pFirst, double* pSecond, double* pOutput, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;

    if (col < k && row < m)
    {
        for (int i = 0; i < n; ++i)
        {
            sum += pFirst[row * n + i] * pSecond[i * k + col];
        }
        pOutput[row * k + col] = sum;
    }
}

void multiply(void* pFirst, void* pSecond, void* pOutput,
              int firstRows, int firstColumns, int secondColumns)
{
    nvtxRangePush(__FUNCTION__);

    unsigned int grid_rows = (firstRows + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (secondColumns + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    multiply_kernel<<<dimGrid, dimBlock>>>((double*)pFirst, (double*)pSecond, (double*)pOutput, firstRows, firstColumns, secondColumns);

    nvtxRangePop();
}

__global__ void multiply_scalar_kernel(double* pOutput, double* pInput, int iLength, double scalar)
{
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < iLength;
         i += blockDim.x * gridDim.x)
    {
        pOutput[i] = pInput[i] * scalar;
    }
}

void multiply_scalar(void* input, void* pOutput, int iLength, double scalar)
{
    multiply_scalar_kernel<<<512, 256>>>((double *) pOutput, (double *) input, iLength, scalar);
}
